#include "hip/hip_runtime.h"
#include "Iteration2.cuh"

void Iteration3(cunfftplan2d2 plan, GICONPara GP,MrcHeader *inhead,int iter_num,int Yvalue,char *resultpath,int *before,float *betain)
{
    clock_t btime, etime;
    int i,N[2],/*n[2],*/beforeupdate,/*threadnum,*/ beforeend,dataType,thickness;
    float threshold;


    beforeupdate = before[0];
    beforeend = before[1];
    //threadnum = before[2];
    dataType = before[3];
    threshold = betain[0];
    thickness = (int)betain[1];


    //float uptop;
    //DATATYPE thita[2];
    int M = inhead->nx*inhead->nz;
    N[0] = plan.N[0];
    N[1] = plan.N[1];
    //uptop = pow(2,ceil(log10(N[0])/log10(2.0)));
    //thita[0] = 2.0*uptop/(double)N[0];
    //uptop = pow(2,ceil(log10(N[1])/log10(2.0)));
    //thita[1] = 2.0*uptop/(double)N[1];

    //n[0] = ceil(thita[0]*N[0]);
    //n[1] = ceil(thita[1]*N[1]);


    btime = clock();
    hipMemset(GP.mCU,0,N[0]*N[1]*sizeof(DATATYPE));
    //DATATYPE * f2 = (DATATYPE*)malloc(inhead->nx*inhead->nx*sizeof(DATATYPE));

    /*hipEvent_t start1;
    hipEventCreate(&start1);
    hipEvent_t stop1;
    hipEventCreate(&stop1);
    float msecTotal1,msecTotal2;*/

    for (i = 0 ; i < iter_num ; i++)
    {
        //Am
	/*{
	DATATYPE * f2 = (DATATYPE*)malloc(N[0]*N[1]*sizeof(DATATYPE));
	hipMemcpy(f2,GP.mCU,N[0]*N[1]*sizeof(DATATYPE),hipMemcpyDeviceToHost);
	for (ii = 0 ; ii < 20 ; ii++)
		printf("f_hat %.20lf\n",f2[ii]);
	free(f2);
	}*/
	//hipEventRecord(start1, NULL);
	cunfft_trafo_2d_R2C2(plan,GP.fCU,GP.mCU);
	/*hipDeviceSynchronize();
   	hipEventRecord(stop1, NULL);
	hipEventSynchronize(stop1);
    	msecTotal1 = 0.0f;
    	hipEventElapsedTime(&msecTotal1, start1, stop1);
    	printf("cunfft_trafo_2d_R2C2 : %f\n ",msecTotal1);*/
	/*{
	CUFFTDATATYPE * f = (CUFFTDATATYPE*)malloc(inhead->nx*inhead->nz*sizeof(CUFFTDATATYPE));
	hipMemcpy(f,GP.fCU,inhead->nx*inhead->nz*sizeof(CUFFTDATATYPE),hipMemcpyDeviceToHost);
	for (ii = 0 ; ii < 20 ; ii++)
		printf("f-- %.20lf %.20lf\n",f[ii].x,f[ii].y);
	free(f);
	}*/
        //wAm
	//hipEventRecord(start1, NULL);
	complexMulDoubleCU<<<dim3(GRIDDIMX,((M - 1)/BLOCKDIM)/GRIDDIMX+1),dim3(BLOCKDIMX,BLOCKDIMY)>>>(GP.fCU,GP.weightCU,M);
	/*hipDeviceSynchronize();
   	hipEventRecord(stop1, NULL);
	hipEventSynchronize(stop1);
    	msecTotal1 = 0.0f;
    	hipEventElapsedTime(&msecTotal1, start1, stop1);
    	printf("complexMulDoubleCU : %f\n ",msecTotal1);*/


        //AhwAm
	//hipEventRecord(start1, NULL);
	cunfft_adjoint_2d2(plan,GP.fCU,GP.fhatCU);
	/*hipDeviceSynchronize();
   	hipEventRecord(stop1, NULL);
	hipEventSynchronize(stop1);
    	msecTotal1 = 0.0f;
    	hipEventElapsedTime(&msecTotal1, start1, stop1);
    	printf("cunfft_adjoint_2d2 : %f\n ",msecTotal1);*/
	/*{
	CUFFTDATATYPE * f = (CUFFTDATATYPE*)malloc(N[0]*N[1]*sizeof(CUFFTDATATYPE));
	hipMemcpy(f,GP.fhatCU,N[0]*N[1]*sizeof(CUFFTDATATYPE),hipMemcpyDeviceToHost);
	for (ii = 0 ; ii < 20 ; ii++)
		printf("ooooo%.20lf %.20lf\n",f[ii].x,f[ii].y);
	free(f);
	}*/

	//hipEventRecord(start1, NULL);
	complexMinusCU<<<dim3(GRIDDIMX,((N[0]*N[1] - 1)/BLOCKDIM)/GRIDDIMX+1),dim3(BLOCKDIMX,BLOCKDIMY)>>>(GP.rhatCU,GP.fhatCU,GP.AhwbCU,N[0]*N[1]);
	/*hipDeviceSynchronize();
   	hipEventRecord(stop1, NULL);
	hipEventSynchronize(stop1);
    	msecTotal1 = 0.0f;
    	hipEventElapsedTime(&msecTotal1, start1, stop1);
    	printf("complexMinusCU : %f\n ",msecTotal1);*/
	/*{	
	CUFFTDATATYPE * f = (CUFFTDATATYPE*)malloc(inhead->nx*inhead->nx*sizeof(CUFFTDATATYPE));
	hipMemcpy(f,GP.rhatCU,inhead->nx*inhead->nx*sizeof(CUFFTDATATYPE),hipMemcpyDeviceToHost);
	for (ii = 0 ; ii < 20 ; ii++)
		printf("rrrrrr%.20lf %.20lf\n",f[ii].x,f[ii].y);
	free(f);
	}*/
	/*hipMemcpy(f,GP.AhwbCU,inhead->nx*inhead->nx*sizeof(CUFFTDATATYPE),hipMemcpyDeviceToHost);
	for (ii = 0 ; ii < 20 ; ii++)
		printf("AAAAA%.20lf %.20lf\n",f[ii].x,f[ii].y);*/

	//hipEventRecord(start1, NULL);
 	Alpha2(inhead,GP,plan,thickness);
	/*hipDeviceSynchronize();
   	hipEventRecord(stop1, NULL);
	hipEventSynchronize(stop1);
    	msecTotal1 = 0.0f;
    	hipEventElapsedTime(&msecTotal1, start1, stop1);
    	printf("Alpha2 : %f\n ",msecTotal1);*/

	/*printf("al %.20lf\n",al);
	hipMemcpy(f,GP.fhatCU,inhead->nx*inhead->nx*sizeof(CUFFTDATATYPE),hipMemcpyDeviceToHost);
	for (ii = 0 ; ii < 20 ; ii++)
		printf("xxx%.20lf %.20lf\n",f[ii].x,f[ii].y);*/
	//printf("threshold %f\n",threshold);
	//hipEventRecord(start1, NULL);
	if (beforeupdate <= i && i < iter_num - beforeend)
		ICONUpdateCU_R<<<dim3(GRIDDIMX,((N[0]*N[1] - 1)/BLOCKDIM)/GRIDDIMX+1),dim3(BLOCKDIMX,BLOCKDIMY)>>>(GP.mCU,GP.rhatCU,GP.alCU,dataType,threshold,N[0]*N[1]);
	else
		INFRUpdateCU_R<<<dim3(GRIDDIMX,((N[0]*N[1] - 1)/BLOCKDIM)/GRIDDIMX+1),dim3(BLOCKDIMX,BLOCKDIMY)>>>(GP.mCU,GP.rhatCU,GP.alCU,N[0]*N[1]);
	/*hipDeviceSynchronize();
   	hipEventRecord(stop1, NULL);
	hipEventSynchronize(stop1);
    	msecTotal1 = 0.0f;
    	hipEventElapsedTime(&msecTotal1, start1, stop1);
    	printf("ICONUpdateCU_R : %f\n ",msecTotal1);*/
	/*hipMemcpy(GP.m,GP.mCU,inhead->nx*inhead->nx*sizeof(DATATYPE),hipMemcpyDeviceToHost);
	for (ii = 0 ; ii < 20 ; ii++)
		printf("%.20lf\n",GP.m[ii]);*/
	/*hipMemcpy(f,GP.mCU,inhead->nx*inhead->nx*sizeof(CUFFTDATATYPE),hipMemcpyDeviceToHost);
	for (ii = 0 ; ii < 20 ; ii++)
		printf("%.20lf %.20lf\n",f[ii].x,f[ii].y);*/

    }
    //hipDeviceSynchronize();
    etime = clock();
    printf("slice %d finished, time %lf\n",Yvalue,(DATATYPE)(etime - btime) / CLOCKS_PER_SEC);
}



