#include "hip/hip_runtime.h"
#include "MainCode_crossV.cuh"

int Lib_2D_crossV2(int sliceBegin,int sliceEnd,int iternum,MrcHeader* inhead,DATATYPE *reprojection,FILE *infile,char *anglefilename,char* resultpath,int *before,float *betain)
{
    char loginfo[1000];
    int Ang_Num,i,N[2],n[2];
    FILE * anglefile;
    float *thital,omitang;
    DATATYPE * weight;
    DATATYPE *bpx2D;
    int Yvalue;
    thital = (float*)malloc(180*sizeof(float));
    GICONPara GP;
    int thickness = (int)betain[1];
    if (thickness == 0)
	thickness = inhead->nx;

    //read mrc head
    MrcHeader *inhead_reProj=(MrcHeader *)malloc(sizeof(MrcHeader));
    MrcHeader *inhead_omit=(MrcHeader *)malloc(sizeof(MrcHeader));
    memcpy(inhead_reProj,inhead,sizeof(MrcHeader));
    memcpy(inhead_omit,inhead,sizeof(MrcHeader));
    inhead_reProj->nz = 1;
    inhead_omit->nz -= 1;
    if ((anglefile = fopen(anglefilename,"r")) == NULL)
    {
        printf("\nCan not open anglefile!\n");
	//log Write
    	sprintf(loginfo,"running state:\n   ");
    	logwrite(loginfo);
    	sprintf(loginfo,"fail!\n");
    	logwrite(loginfo);
	sprintf(loginfo,"Error message:\n   ");
	logwrite(loginfo);
	sprintf(loginfo,"Can not open anglefile '%s'!\n",anglefilename);
	logwrite(loginfo);
    	//end of log Write	
        exit(1);
    }

    Ang_Num = 0;
    while (fscanf(anglefile,"%f",&(thital[Ang_Num]))!=EOF)
    {
	// simplest way to generate reconstruction as IMOD(xyz)
        thital[Ang_Num] *=-1;
        Ang_Num++;
    }
    fclose(anglefile);
    omitang = thital[0];
    int omitindex = 0;
    for (i = 0 ; i < Ang_Num ; i++)
	if (fabs(omitang) > fabs(thital[i]))
	{
		omitang = thital[i];
		omitindex = i;
	}
    Ang_Num--;
    for (i = omitindex ; i < Ang_Num ; i++)
	thital[i] = thital[i+1];
    //end of read projection
  
    weight = (DATATYPE *)malloc(inhead_omit->nx*inhead_omit->nz*sizeof(DATATYPE));
    Weight(weight,inhead_omit,thital);
  
    initGICONPara2(&GP,inhead_omit->nx,thickness,inhead_omit->nz,weight);

    bpx2D = (DATATYPE*)malloc(2*inhead_omit->nx*inhead_omit->nz*sizeof(DATATYPE));
    px2D(bpx2D,inhead_omit,thital);


	float uptop;
    DATATYPE thita[2];
    int nfft_m = NFFTM;
    int M = inhead_omit->nx*inhead_omit->nz;
    N[0] = inhead_omit->nx;
    N[1] = thickness;
    uptop = pow(2,ceil(log10(N[0])/log10(2.0)));
    thita[0] = 2.0*uptop/(double)N[0];
    uptop = pow(2,ceil(log10(N[1])/log10(2.0)));
    thita[1] = 2.0*uptop/(double)N[1];
    n[0] = ceil(thita[0]*N[0]);
    n[1] = ceil(thita[1]*N[1]);
    cunfftplan2d2 plan2d;
    DATATYPE * ck2dCU; 
    CUFFTDATATYPE *g2dCU;
    hipfftHandle planTmp2d;
    /*for (int ms = 0 ; ms < 100000000;ms++)
	ms = 1000*1000;*/
    hipMalloc((void**)&(ck2dCU), sizeof(DATATYPE)*N[0]*N[1]);
    /*for (int ms = 0 ; ms < 100000000;ms++)
	ms = 1000*1000;*/
    hipMalloc((void**)&(g2dCU), sizeof(CUFFTDATATYPE)*n[0]*n[1]);  
    hipfftPlan2d(&(planTmp2d),n[0],n[1],CUFFTTYPE); 
    cunfft_mallocPlan_2d2(&plan2d,N[0],N[1],thita,M,nfft_m,CUNFFTPLAN_NO_MALLOC_CK | CUNFFTPLAN_NO_MALLOC_G | CUNFFTPLAN_NO_MALLOC_PLAN| CUNFFTPLAN_NO_PRECOMPUTE);
    plan2d.ckCU = ck2dCU;
    plan2d.gCU = g2dCU;
    plan2d.planTmp = planTmp2d;
    cunfft_initPlan_2d2(&plan2d,N[0],N[1],thita,M,nfft_m,bpx2D);
    DATATYPE *px1d;
    px1d = (DATATYPE*)malloc(inhead_omit->nx*sizeof(DATATYPE));
    int k;
    for (k = 0 ; k < inhead_omit->nx ; k++)
    {
        px1d[k] = ((DATATYPE)k-(DATATYPE)inhead_omit->nx/2.0)/(DATATYPE)inhead_omit->nx;
    }
    cunfftplan1d plan1d;
    cunfft_initPlan_1d(&plan1d,N[0],thita[0],N[0],nfft_m,px1d);
    /*DATATYPE *ck1d = (DATATYPE *)malloc(N[0]*sizeof(DATATYPE));
    hipMemcpy(ck1d,plan1d.ckCU,N[0]*sizeof(DATATYPE),hipMemcpyDeviceToHost);
    for (i = 0 ; i < 20 ; i++)
	printf("%.20lf\n",ck1d[i]);
    free(ck1d);*/
    free(px1d);
    float thital2[1];
    thital2[0] = omitang;
    px2D(bpx2D,inhead_reProj,thital2);
    cunfftplan2d2 plan2d_reProj;
    cunfft_mallocPlan_2d2(&plan2d_reProj,N[0],N[1],thita,N[0],nfft_m,CUNFFTPLAN_NO_MALLOC_CK | CUNFFTPLAN_NO_MALLOC_G | CUNFFTPLAN_NO_MALLOC_PLAN| CUNFFTPLAN_NO_PRECOMPUTE);
    plan2d_reProj.ckCU = ck2dCU;
    plan2d_reProj.gCU = g2dCU;
    plan2d_reProj.planTmp = planTmp2d;    
    cunfft_initPlan_2d2(&plan2d_reProj,N[0],N[1],thita,N[0],nfft_m,bpx2D);
    //CUFFTDATATYPE * f = (CUFFTDATATYPE*)malloc(inhead->nx*inhead->nx*sizeof(CUFFTDATATYPE));
    //int ii;
    for (Yvalue = sliceBegin ; Yvalue < sliceEnd && Yvalue < inhead_omit->ny; Yvalue++)
    {
        readSlicePart_crossV(plan1d,GP.fhat1dCU,GP.fCU,infile,inhead,Yvalue,omitindex);
	/*hipMemcpy(f,GP.fCU,inhead->nx*inhead->nz*sizeof(CUFFTDATATYPE),hipMemcpyDeviceToHost);
	for (ii = 0 ; ii < 20 ; ii++)
		printf("%f %f\n",f[ii].x,f[ii].y);
	free(f);
	return 0;	*/

    //wb
	complexMulDoubleCU<<<dim3(GRIDDIMX,((M - 1)/BLOCKDIM)/GRIDDIMX+1),dim3(BLOCKDIMX,BLOCKDIMY)>>>(GP.fCU,GP.weightCU,M);
	/*hipMemcpy(f,GP.fCU,inhead->nx*inhead->nz*sizeof(CUFFTDATATYPE),hipMemcpyDeviceToHost);
	for (ii = 0 ; ii < 20 ; ii++)
		printf("yyyyyyy%.20lf %.20lf\n",f[ii].x,f[ii].y);*/
    //Ahwb
	cunfft_adjoint_2d2(plan2d,GP.fCU,GP.fhatCU);
	/*hipMemcpy(f,GP.fhatCU,N[0]*N[1]*sizeof(CUFFTDATATYPE),hipMemcpyDeviceToHost);
	for (ii = 0 ; ii < 20 ; ii++)
		printf("%.20lf %.20lf\n",f[ii].x,f[ii].y);
	free(f);
	return 0;*/
	complexCopyCU<<<dim3(GRIDDIMX,((N[0]*N[1] - 1)/BLOCKDIM)/GRIDDIMX+1),dim3(BLOCKDIMX,BLOCKDIMY)>>>(GP.AhwbCU,GP.fhatCU,N[0]*N[1]);

        Iteration3(plan2d,GP,inhead_omit,iternum,Yvalue,resultpath,before,betain);
	hipMemcpy(GP.m,GP.mCU,N[0]*N[1]*sizeof(DATATYPE),hipMemcpyDeviceToHost);
	/*for (i = 0 ; i < 20 ; i++)
		printf("########%.20lf %.20lf\n",GP.m[i].x,GP.m[i].y);*/
	printf("cross validation!\n");
	reProject_NFFT2(plan1d,plan2d_reProj,GP,reprojection+(Yvalue-sliceBegin)*inhead_omit->nx);
	/*for (i = 0 ; i < 20 ; i++)
		printf("----------%.20lf %.20lf\n",reprojection[i].x,reprojection[i].y);*/
	{
		//FILE * fff = fopen("txt","w+");
		/*char __fd_name[500];
    		char resultpath2[1000];
    		char outfilename[1000];

    		strcpy(resultpath2,resultpath);
    		sprintf(outfilename,"/mid%05d.mrc",Yvalue);
    		strcat(resultpath2,outfilename);
   		strcpy(outfilename,resultpath2);

    		FILE *outfile = fopen(outfilename,"w+");
		printf("%s\n",outfilename);
   		MrcHeader *outhead = (MrcHeader *)malloc(sizeof(MrcHeader));
		saveMRC_real_inv(inhead,outfile,outhead,GP.m,inhead_omit->nx,thickness);
    		free(outhead);
    		fclose(outfile);
    		mrc_update_head(outfilename);*/
	}
    }
    cunfft_destroyPlan_1d(&plan1d);
    cunfft_destroyPlan_2d2(&plan2d);
    cunfft_destroyPlan_2d2(&plan2d_reProj);
    hipFree(ck2dCU);
    hipFree(g2dCU);
    hipfftDestroy(planTmp2d);
    destroyGICONPara(&GP);
    free(bpx2D);
    free(weight);
    free(inhead_reProj);
    free(inhead_omit);
    free(thital);
    return 0;
}

